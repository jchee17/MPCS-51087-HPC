#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NDIM 5

__global__ void init_rand(hiprandState *state)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1, index, 0, &state[index]);
}

/* I believe hiprand_uniform gen between 0 and 1 */
/* this code isn't code right, yet */
__global__ void gen_rand(hiprandState *state, double *randArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int i;
	for (i = index; i < 5*n; i += nblocks*nthreads)
		randArray[i] = hiprand_uniform(&state[index]);
}

/* implement local array to save */
__global__ void eval_function(double *randArray, double *int_f, int *n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int start = 5 * index;

	double f = exp(-randArray[start]*randArray[start] 
			-randArray[start+1]*randArray[start+1] 
			-randArray[start+2]*randArray[start+2]  
			-randArray[start+3]*randArray[start+3] 
			-randArray[start+4]*randArray[start+4]);
	int_f[index] = f / (*n);
}

__global__ void reduction(double *int_f, double *int_f_master, int *n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index == 0)
	{
		*int_f_master = 0.0;
		int i;
		for (i = 0; i < *n; i++)
			*int_f_master += int_f[i];
	}
}

int main(int argc, char **argv)
{
	int n, *d_n;
	double t0, t1;
	double vol=1.0;

	int nBlocks  = 2;
	int nThreads = 64;

	n = atoi(argv[1]);

	hipMalloc(&d_n, sizeof(int));
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	hiprandState *d_state;
	hipMalloc(&d_state, nThreads * nBlocks);

	double *randArray;
	hipMalloc(&randArray, 5 * n * sizeof(double));

	double *int_f;
	hipMalloc(&int_f, n * sizeof(double));

	double *int_f_master;
	hipMalloc(&int_f_master, sizeof(double));

	init_rand    <<<nThreads, nBlocks>>>( d_state );
	gen_rand     <<<nThreads, nBlocks>>>( d_state, randArray);
	eval_function<<<nThreads, nBlocks>>>( randArray, int_f, d_n);
	reduction    <<<nThreads, nBlocks>>>( int_f, int_f_master, d_n);

	double integral;
	hipMemcpy(&integral, int_f_master, sizeof(double), 
		hipMemcpyDeviceToHost);

	//int i;
	//for (i = 0; i < NDIM; i++)
	//	vol *= (b-a);

	printf("%lf\n", integral);

	hipFree(d_state);
	hipFree(randArray);
	hipFree(int_f);
	hipFree(int_f_master);	
}
