#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void random_test(hiprandState *state, float *randArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1, index, 0, &state[index]);

	randArray[index] = hiprand_uniform(&state[index]);
}

int main(int argc, char* argv[])
{
	int nblocks;
	int nthreads;

	nblocks = atoi(argv[1]);
	nthreads = atoi(argv[2]);

	hiprandState *d_state;
	hipMalloc(&d_state, nthreads * nblocks);

	float *randArray;
	hipMalloc(&randArray, nblocks*nthreads*sizeof(float));

	random_test<<<nthreads, nblocks>>>( d_state, randArray);

	float *rand_loc;
	rand_loc = (float *) malloc(nblocks * nthreads * sizeof(float));

	hipMemcpy(rand_loc, randArray, nthreads*nblocks*sizeof(float),
			hipMemcpyDeviceToHost);

	int i;
	for (i = 0; i < nblocks*nthreads; i++)
		printf("%f\n", rand_loc[i]);

	hipFree(d_state);
	hipFree(randArray);
	free(rand_loc);

}
